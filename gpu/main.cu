#include "hip/hip_runtime.h"
#include <SDL2/SDL.h>
#include <iostream>
#include <string.h>
#include <math.h>

// Screen size
#define SCREEN_HEIGHT (700)
#define SCREEN_WIDTH  (700)

// Max number of iteration for the check of convergence
#define MAX_ITERATION (100)

// Window of the complex plane to display
#define W1X (-1.5)
#define W1Y (-1.1)
#define W2X (0.5)
#define W2Y (1.1)

// Size of the buffer containing the image of the Mandelbrot set
#define BUFFER_SIZE  (SCREEN_HEIGHT * SCREEN_WIDTH * 4)
#define CBUFFER_SIZE (BUFFER_SIZE * sizeof(char))

typedef struct
{
    float x;
    float y;
} complex;

__device__ float modulus(complex z) {
    return sqrt(pow(z.x, 2) + pow(z.y, 2));
}

__device__ complex add(complex z0, complex z1) {
    complex res = {0};

    res.x = z0.x + z1.x;
    res.y = z0.y + z1.y;
    return res;
}

__device__ complex mul(complex z0, complex z1) {
    complex res = {0};

    res.x = z0.x * z1.x - z0.y * z1.y;
    res.y = z0.x * z1.y + z0.y * z1.x;
    return res;
}

__global__ void mandelbrot_set(char *data)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id  = 4 * (idx + idy * SCREEN_WIDTH);

    if (idx >= SCREEN_WIDTH || idy >= SCREEN_HEIGHT) {
        return;
    }
    complex z0 = {0};
    complex c = {0};
    c.x = W1X + idx * (W2X - W1X) / SCREEN_WIDTH;
    c.y = W2Y - idy * (W2Y - W1Y) / SCREEN_HEIGHT;
    for (int i = 0; i < MAX_ITERATION; ++i) {
        z0 = add(mul(z0, z0), c);
        if (modulus(z0) > 2) {
            data[id]     = 255;
            data[id + 1] = 255;
            data[id + 2] = 255 * i / MAX_ITERATION;
            data[id + 3] = 255 * i / MAX_ITERATION;
            return;
        }
    }

}

SDL_Surface *compute_mandelbrot_set() {

    // Check if at least one CUDA device is available
    int devCount;
    hipGetDeviceCount(&devCount);
    hipError_t err = hipGetDeviceCount(&devCount);
    if (err != hipSuccess) {
        printf("%s\n", hipGetErrorString(err));
        exit(-1);
    }
    if (devCount <= 0) {
        printf("No CUDA gpu available on this system.\n");
        exit(-1);
    }

    // Allocate the data and run the kernel
    char *data;
    hipMallocManaged(&data, CBUFFER_SIZE);
    dim3 block(20,20, 1);
    dim3 grid(35, 35, 1);
    mandelbrot_set<<<grid,block>>>(data);
    hipDeviceSynchronize();

    void *hostData = malloc(CBUFFER_SIZE);
    hipMemcpy(hostData, data, CBUFFER_SIZE, hipMemcpyDeviceToHost);

    // Copy result in the SDL surface
    SDL_Surface *image = SDL_CreateRGBSurfaceWithFormatFrom(hostData,
                                                            SCREEN_WIDTH,
                                                            SCREEN_HEIGHT,
                                                            32,
                                                            SCREEN_WIDTH * 4,
                                                            SDL_PIXELFORMAT_RGBA8888);
    hipFree(data);
    return image;
}

int main() {

    if(SDL_Init(SDL_INIT_VIDEO) < 0)
    {
        std::cout << "Failed to initialize the SDL2 library\n";
        return -1;
    }

    SDL_Window *window = SDL_CreateWindow("Mandelbrot set",
                                          SDL_WINDOWPOS_CENTERED,
                                          SDL_WINDOWPOS_CENTERED,
                                          SCREEN_HEIGHT,
                                          SCREEN_WIDTH,
                                          0);

    if(!window)
    {
        std::cout << "Failed to create window\n";
        return -1;
    }

    SDL_Surface *window_surface = SDL_GetWindowSurface(window);

    if(!window_surface)
    {
        std::cout << "Failed to get the surface from the window\n";
        return -1;
    }

    SDL_Surface *image = compute_mandelbrot_set();
    SDL_BlitSurface(image, NULL, window_surface, NULL);
    SDL_UpdateWindowSurface(window);

    bool keep_window_open = true;
    while(keep_window_open)
    {
        SDL_Event e;
        while(SDL_PollEvent(&e) > 0)
        {
            switch(e.type)
            {
                case SDL_QUIT:
                    keep_window_open = false;
                    break;
            }
        }
    }
}
